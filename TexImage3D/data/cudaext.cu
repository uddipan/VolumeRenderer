#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES
#include<stdio.h>
#include <cutil_inline.h>
#include <cutil_math.h>
#include<iostream>
#include<cuda_gl_interop.h>
#include "hip/hip_runtime.h"
#include ""
#include "cuPrintf.cu"
#include<vector>

typedef unsigned int  uint;
typedef unsigned char uchar;
using namespace std;

typedef struct {
	float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix


struct HIT{
	float n,f;
};

//struct for defining a ray
struct RAY{
	float3 origin,direction;
};

hipArray *d_volumeArray = 0;
hipArray* d_transferFuncArray;
hipArray* posarray;



texture<uchar,  3, hipReadModeNormalizedFloat> tex2[2];

texture<uchar,  3, hipReadModeNormalizedFloat> tex;         // 3D texture for the volume
//texture<float4, 1, hipReadModeElementType> transferTex; // 1D transfer function texture for color
texture<float4, 1, hipReadModeElementType> transferTex; // 1D transfer function texture for color
texture<float, 1, hipReadModeElementType> key_pos; // 1D transfer function texture for key pos

/*----------------------------------------------------------------------------
copy volume data into 3D texture memory and 
set up the transfer function texture memory
-----------------------------------------------------------------------------*/
void initCuda(uchar *volume, hipExtent volumeSize){
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
	cutilSafeCall( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );

	// copy data to 3D array
	hipMemcpy3DParms copyParams = {0};
	copyParams.srcPtr   = make_hipPitchedPtr((void*)volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent   = volumeSize;
	copyParams.kind     = hipMemcpyHostToDevice;
	cutilSafeCall( hipMemcpy3D(&copyParams) );  

	// set texture parameters
	tex.normalized = true;                      // access with normalized texture coordinates
	tex.filterMode = hipFilterModeLinear;      // linear interpolation
	tex.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates
	tex.addressMode[1] = hipAddressModeClamp;

	// bind array to 3D texture
	cutilSafeCall(hipBindTextureToArray(tex, d_volumeArray, channelDesc));
}


//the color is put on a texture memory
void initCuda_color(vector<float4> a){
	float4 *transferFunc;
	transferFunc=(float4*)malloc(a.size()*4*sizeof(float));
	copy( a.begin(), a.end(), transferFunc);


	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	cutilSafeCall(hipMallocArray( &d_transferFuncArray, &channelDesc2, a.size(), 1)); 
	cutilSafeCall(hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, a.size()*4*sizeof(float), hipMemcpyHostToDevice));



	transferTex.filterMode = hipFilterModeLinear;
	transferTex.normalized = true;    // access with normalized texture coordinates
	transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	// Bind the array to the texture
	cutilSafeCall( hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc2));
	delete transferFunc;
}


/*------------------------------------------------------------------------------------------------
set the key positions in the global memory
------------------------------------------------------------------------------------------------*/
void set_key_position(vector<float> keys){
	float *knots=(float*)malloc(keys.size()*sizeof(float));
	copy(keys.begin(),keys.end(),knots);


	hipChannelFormatDesc channelDesc3 = hipCreateChannelDesc<float>();
	cutilSafeCall(hipMallocArray( &posarray, &channelDesc3, keys.size(), 1)); 
	cutilSafeCall(hipMemcpyToArray( posarray, 0, 0, knots, keys.size()*sizeof(float), hipMemcpyHostToDevice));

	key_pos.filterMode = hipFilterModeLinear;
	key_pos.normalized = true;    // access with normalized texture coordinates
	key_pos.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

	// Bind the array to the texture
	cutilSafeCall( hipBindTextureToArray( key_pos, posarray, channelDesc3));
	delete knots;
}
/*------------------------------------------------------------------------------------------------
interpolation mode
------------------------------------------------------------------------------------------------*/
void setTextureFilterMode(bool bLinearFilter){
	tex.filterMode = bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
}
/*------------------------------------------------------------------------------------------------
free up all CUDA buffers
------------------------------------------------------------------------------------------------*/
void freeCudaBuffers()
{
	cutilSafeCall(hipFreeArray(d_volumeArray));
	cutilSafeCall(hipFreeArray(d_transferFuncArray));
	cutilSafeCall(hipFreeArray(posarray));
}

/*------------------------------------------------------------------------------------------------
free up color buffers
------------------------------------------------------------------------------------------------*/
void freecolorbuffer(){
	cutilSafeCall(hipFreeArray(d_transferFuncArray));
	cutilSafeCall(hipFreeArray(posarray));
}
/*------------------------------------------------------------------------------------------------
copy the model view matrix from OpenGL into the device memory.
This is required while defining the origin and direction of the ray
-------------------------------------------------------------------------------------------------*/
void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix)
{
	cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix) );
}

/*------------------------------------------------------------------------------------------------
transformation of a vector by amatrix
-------------------------------------------------------------------------------------------------*/
__device__ float4 mul(const float3x4 &M, const float4 &v){
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}
__device__ float3 mul(const float3x4 &M, const float3 &v){
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}
/*------------------------------------------------------------------------------------------------
assign RGBA values
-------------------------------------------------------------------------------------------------*/
__device__ uint rgbaFloatToInt(float4 rgba){
	rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(rgba.y);
	rgba.z = __saturatef(rgba.z);
	rgba.w = __saturatef(rgba.w);
	return (uint(rgba.w*255)<<24) | (uint(rgba.z*255)<<16) | (uint(rgba.y*255)<<8) | uint(rgba.x*255);
}
/*------------------------------------------------------------------------------------------------
RAY BOX INTERSECTION
-------------------------------------------------------------------------------------------------*/
__device__ HIT find_hit(RAY R,float3 boxmin,float3 boxmax){
	HIT H;H.n=0;H.f=0;
	float N,F ;//=-1000.0f,F=-near;//initialize the hit
	N=-100;F=100;
	if(R.direction.x==0){//ray parallel to X plane
		if((R.origin.x<boxmin.x)||(R.origin.x>boxmax.x))
			return H;
	}
	else{
		//compute the intersection distance of the planes
		float T1=(boxmin.x-R.origin.x)/R.direction.x;
		float T2=(boxmax.x-R.origin.x)/R.direction.x;
		if(T1>T2){//swap
			T1=T1+T2;T2=T1-T2;T1=T1-T2;
		}
		if(T1>N)N=T1;
		if(T2<F)F=T2;
	}

	//check with Y planes
	if(R.direction.y==0){//ray parallel to X plane
		if((R.origin.y<boxmin.y)||(R.origin.y>boxmax.y))
			return H;
	}
	else{
		//compute the intersection distance of the planes
		float T1=(boxmin.y-R.origin.y)/R.direction.y;
		float T2=(boxmax.y-R.origin.y)/R.direction.y;
		if(T1>T2){//swap
			T1=T1+T2;T2=T1-T2;T1=T1-T2;
		}
		if(T1>N)N=T1;
		if(T2<F)F=T2;
	}

	//check with Z planes
	if(R.direction.z==0){//ray parallel to X plane
		if((R.origin.z<boxmin.z)||(R.origin.z>boxmax.z))
			return H;
	}
	else{
		//compute the intersection distance of the planes
		float T1=(boxmin.z-R.origin.z)/R.direction.z;
		float T2=(boxmax.z-R.origin.z)/R.direction.z;

		if(T1>T2){//swap
			T1=T1+T2;T2=T1-T2;T1=T1-T2;
		}
		if(T1>N)N=T1;
		if(T2<F)F=T2;
	}

	if(N>F)return H;//box is missed
	if(F<0)return H;//box behind the ray

	H.n=N;H.f=F;
	return H;

}

/*------------------------------------------------------------------------------------------------
MAIN KERNEL
-------------------------------------------------------------------------------------------------*/
__global__ void kernel(uint *d_output,uint width, uint height,float density,float brightness,float projection){
	int no_of_samples=500;//no of samples between near and far hit
	float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);//the bbox of the volume
	float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	int x=threadIdx.x+blockIdx.x*blockDim.x;
	int y=threadIdx.y+blockIdx.y*blockDim.y;
	if ((x >= width) || (y >= height)) return;

	float u = (x / (float) width)*2.0f-1.0f;
	float v = (y / (float) height)*2.0f-1.0f;
	RAY ray;
	if(projection==0){
		//	perspective projection
		ray.origin=make_float3(mul(c_invViewMatrix,make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
		ray.direction = normalize(make_float3(u, v, -2.0f));
		ray.direction = mul(c_invViewMatrix, ray.direction);
	}
	else{
		//orthographic projection used here
		ray.origin=make_float3(mul(c_invViewMatrix,make_float4(u*projection,v*projection, 0.0f, 1.0f)));
		ray.direction = normalize(make_float3(0.0f,0.0f,- 2.0f));
		ray.direction = mul(c_invViewMatrix, ray.direction);
	}

	HIT H;
	H=find_hit(ray,boxMin,boxMax);
	//if there is no hit
	if((H.n==0)&&(H.f==0)) return;
	if (H.n < 0.0f) H.n = 0.0f;     // clamp to near plane
	//now ray marching thru the volume from back to front



	float4 sum=make_float4(0.0f);
	float t=H.f,tstep=0.01;
	for(int i=0;i<no_of_samples;i++){
		//position along the ray
		float3 pos=ray.origin+ray.direction*t;
		pos = pos*0.5f+0.5f;    // map position to [0, 1] coordinates
		// read from 3D texture
		float sample = tex3D(tex, pos.x, pos.y, pos.z);
		// lookup in transfer function texture
		sample=tex1D(key_pos,sample);
		float4 col = tex1D(transferTex, sample/256.0f);
		// accumulate result
		sum = lerp(sum, col, col.w*density);
		t -= tstep;
		if (t < H.n) break;
	}

	sum *= brightness;

	if ((x < width) && (y < height)) {
		// write output color
		uint i = y*width + x;
		d_output[i] = rgbaFloatToInt(sum);
	}
}


void render_kernel(dim3 grids, dim3 threads, uint *d_output, uint width,uint height, float density, 
	float brightness,float projection){
		cudaPrintfInit();
		kernel<<<grids,threads>>>(d_output,width,height,density,brightness,projection);
		cudaPrintfDisplay(stdout, true);
		cudaPrintfEnd();

}